#include "hip/hip_runtime.h"
#include "cuda_helpers.cuh"

#define float3 Float3
struct Float3
{
  float x,y,z;
  __device__ friend Float3 operator+(const Float3 a, const Float3 b)
  {
    Float3 c;
    c.x = a.x+b.x;
    c.y = a.y+b.y;
    c.z = a.z+b.z;
    return c;
  }
  __device__ friend Float3 operator-(const Float3 a, const Float3 b)
  {
    Float3 c;
    c.x = a.x-b.x;
    c.y = a.y-b.y;
    c.z = a.z-b.z;
    return c;
  }
  __device__ friend Float3 operator/(const Float3 a, const Float3 b)
  {
    Float3 c;
    c.x = a.x/b.x;
    c.y = a.y/b.y;
    c.z = a.z/b.z;
    return c;
  }
  __device__ friend Float3 operator/(const float a, const Float3 b)
  {
    Float3 c;
    c.x = a/b.x;
    c.y = a/b.y;
    c.z = a/b.z;
    return c;
  }
  __device__ friend Float3 operator*(const Float3 a, const Float3 b)
  {
    Float3 c;
    c.x = a.x*b.x;
    c.y = a.y*b.y;
    c.z = a.z*b.z;
    return c;
  }
  __device__ friend Float3 operator*(const Float3 a, const float b)
  {
    Float3 c;
    c.x = a.x*b;
    c.y = a.y*b;
    c.z = a.z*b;
    return c;
  }
};

#define int8 char
#define int16 short

struct Ray {
    float3 origin, dir, invDir;
    unsigned int dirIsNeg0, dirIsNeg1, dirIsNeg2;
    float mint, maxt;
    int hitId;
};

struct Triangle {
    float p[3][4];
    int id;
    int pad[3];
};

struct LinearBVHNode {
    float bounds[2][3];
    unsigned int offset;     // num primitives for leaf, second child for interior
    unsigned int8 nPrimitives;
    unsigned int8 splitAxis;
    unsigned int16 pad;
};

__device__
static inline float3 Cross(const float3 v1, const float3 v2) {
    float v1x = v1.x, v1y = v1.y, v1z = v1.z;
    float v2x = v2.x, v2y = v2.y, v2z = v2.z;
    float3 ret;
    ret.x = (v1y * v2z) - (v1z * v2y);
    ret.y = (v1z * v2x) - (v1x * v2z);
    ret.z = (v1x * v2y) - (v1y * v2x);
    return ret;
}

__device__
static inline float Dot(const float3 a, const float3 b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__
inline
static void generateRay( const float raster2camera[4][4],
                         const float camera2world[4][4],
                        float x, float y, Ray &ray) {
    ray.mint = 0.f;
    ray.maxt = 1e30f;

    ray.hitId = 0;

    // transform raster coordinate (x, y, 0) to camera space
    float camx = raster2camera[0][0] * x + raster2camera[0][1] * y + raster2camera[0][3];
    float camy = raster2camera[1][0] * x + raster2camera[1][1] * y + raster2camera[1][3];
    float camz = raster2camera[2][3];
    float camw = raster2camera[3][3];
    camx /= camw;
    camy /= camw;
    camz /= camw;

    ray.dir.x = camera2world[0][0] * camx + camera2world[0][1] * camy +
        camera2world[0][2] * camz;
    ray.dir.y = camera2world[1][0] * camx + camera2world[1][1] * camy +
        camera2world[1][2] * camz;
    ray.dir.z = camera2world[2][0] * camx + camera2world[2][1] * camy +
        camera2world[2][2] * camz;

    ray.origin.x = camera2world[0][3] / camera2world[3][3];
    ray.origin.y = camera2world[1][3] / camera2world[3][3];
    ray.origin.z = camera2world[2][3] / camera2world[3][3];

    ray.invDir = 1.f / ray.dir;

#if 0
    ray.dirIsNeg[0] = any(ray.invDir.x < 0) ? 1 : 0;
    ray.dirIsNeg[1] = any(ray.invDir.y < 0) ? 1 : 0;
    ray.dirIsNeg[2] = any(ray.invDir.z < 0) ? 1 : 0;
#else
    ray.dirIsNeg0 = any(ray.invDir.x < 0) ? 1 : 0;
    ray.dirIsNeg1 = any(ray.invDir.y < 0) ? 1 : 0;
    ray.dirIsNeg2 = any(ray.invDir.z < 0) ? 1 : 0;
#endif
}

__device__
inline
static bool BBoxIntersect(const  float bounds[2][3],
                          const Ray &ray) {
     float3 bounds0 = { bounds[0][0], bounds[0][1], bounds[0][2] };
     float3 bounds1 = { bounds[1][0], bounds[1][1], bounds[1][2] };
    float t0 = ray.mint, t1 = ray.maxt;

    // Check all three axis-aligned slabs.  Don't try to early out; it's
    // not worth the trouble
    float3 tNear = (bounds0 - ray.origin) * ray.invDir;
    float3 tFar  = (bounds1 - ray.origin) * ray.invDir;
    if (tNear.x > tFar.x) {
        float tmp = tNear.x;
        tNear.x = tFar.x;
        tFar.x = tmp;
    }
    t0 = max(tNear.x, t0);
    t1 = min(tFar.x, t1);

    if (tNear.y > tFar.y) {
        float tmp = tNear.y;
        tNear.y = tFar.y;
        tFar.y = tmp;
    }
    t0 = max(tNear.y, t0);
    t1 = min(tFar.y, t1);

    if (tNear.z > tFar.z) {
        float tmp = tNear.z;
        tNear.z = tFar.z;
        tFar.z = tmp;
    }
    t0 = max(tNear.z, t0);
    t1 = min(tFar.z, t1);

    return (t0 <= t1);
}


__device__
inline
static bool TriIntersect(const  Triangle &tri, Ray &ray) {
     float3 p0 = { tri.p[0][0], tri.p[0][1], tri.p[0][2] };
     float3 p1 = { tri.p[1][0], tri.p[1][1], tri.p[1][2] };
     float3 p2 = { tri.p[2][0], tri.p[2][1], tri.p[2][2] };
     float3 e1 = p1 - p0;
     float3 e2 = p2 - p0;

    float3 s1 = Cross(ray.dir, e2);
    float divisor = Dot(s1, e1);
    bool hit = true;

    if (divisor == 0.)
        hit = false;
    float invDivisor = 1.f / divisor;

    // Compute first barycentric coordinate
    float3 d = ray.origin - p0;
    float b1 = Dot(d, s1) * invDivisor;
    if (b1 < 0. || b1 > 1.)
        hit = false;

    // Compute second barycentric coordinate
    float3 s2 = Cross(d, e1);
    float b2 = Dot(ray.dir, s2) * invDivisor;
    if (b2 < 0. || b1 + b2 > 1.)
        hit = false;

    // Compute _t_ to intersection point
    float t = Dot(e2, s2) * invDivisor;
    if (t < ray.mint || t > ray.maxt)
        hit = false;

    if (hit) {
        ray.maxt = t;
        ray.hitId = tri.id;
    }
    return hit;
}

__device__
inline
bool BVHIntersect(const  LinearBVHNode nodes[],
                  const  Triangle tris[], Ray &r,
                   int todo[]) {
    Ray ray = r;
    bool hit = false;
    // Follow ray through BVH nodes to find primitive intersections
     int todoOffset = 0, nodeNum = 0;

    while (true) {
        // Check ray against BVH node
         LinearBVHNode node = nodes[nodeNum];
        if (any(BBoxIntersect(node.bounds, ray))) {
             unsigned int nPrimitives = node.nPrimitives;
            if (nPrimitives > 0) {
                // Intersect ray with primitives in leaf BVH node
                 unsigned int primitivesOffset = node.offset;
                for ( unsigned int i = 0; i < nPrimitives; ++i) {
                    if (TriIntersect(tris[primitivesOffset+i], ray))
                        hit = true;
                }
                if (todoOffset == 0)
                    break;
                nodeNum = todo[--todoOffset];
            }
            else {
                // Put far BVH node on _todo_ stack, advance to near node
                int dirIsNeg;
                if (node.splitAxis == 0) dirIsNeg = r.dirIsNeg0;
                if (node.splitAxis == 1) dirIsNeg = r.dirIsNeg1;
                if (node.splitAxis == 2) dirIsNeg = r.dirIsNeg2;
                if (dirIsNeg) {
                   todo[todoOffset++] = nodeNum + 1;
                   nodeNum = node.offset;
                }
                else {
                   todo[todoOffset++] = node.offset;
                   nodeNum = nodeNum + 1;
                }
            }
        }
        else {
            if (todoOffset == 0)
                break;
            nodeNum = todo[--todoOffset];
        }
    }
    r.maxt = ray.maxt;
    r.hitId = ray.hitId;

    return hit;
}

__device__
inline
static void raytrace_tile( int x0,  int x1,
                           int y0,  int y1,
                           int width,  int height,
                           int baseWidth,  int baseHeight,
                          const  float raster2camera[4][4],
                          const  float camera2world[4][4],
                           float image[],  int id[],
                          const  LinearBVHNode nodes[],
                          const  Triangle triangles[]) {
     float widthScale = (float)(baseWidth) / (float)(width);
     float heightScale = (float)(baseHeight) / (float)(height);

#if 0
   int *  todo =  new  int[64];
#define ALLOC
#else
   int todo[64];
#endif

    for (int y = y0 ;y < y1; y++)
      for (int x = x0 + programIndex; x < x1; x += programCount)
        if (x < x1)
        {
          Ray ray;
          generateRay(raster2camera, camera2world, x*widthScale,
              y*heightScale, ray);
          BVHIntersect(nodes, triangles, ray, todo);

          int offset = y * width + x;
          image[offset] = ray.maxt;
          id[offset] = ray.hitId;
        }

#ifdef ALLOC
  delete todo;
#endif
}



__global__
void raytrace_tile_task( int width,  int height,
                              int baseWidth,  int baseHeight,
                             const  float raster2camera[4][4],
                             const  float camera2world[4][4],
                              float image[],  int id[],
                             const  LinearBVHNode nodes[],
                             const  Triangle triangles[]) {
     int dx = 64, dy = 8; // must match dx, dy below
     int xBuckets = (width + (dx-1)) / dx;
     int x0 = (taskIndex % xBuckets) * dx;
     int x1 = min(x0 + dx, width);
     int y0 = (taskIndex / xBuckets) * dy;
     int y1 = min(y0 + dy, height);

    raytrace_tile(x0, x1, y0, y1, width, height, baseWidth, baseHeight,
                  raster2camera, camera2world, image,
                  id, nodes, triangles);
}


extern "C" __global__ void raytrace_ispc_tasks___export( int width,  int height,
                                 int baseWidth,  int baseHeight,
                                const  float raster2camera[4][4],
                                const  float camera2world[4][4],
                                 float image[],  int id[],
                                const  LinearBVHNode nodes[],
                                const  Triangle triangles[]) {
     int dx = 64, dy = 8;
     int xBuckets = (width + (dx-1)) / dx;
     int yBuckets = (height + (dy-1)) / dy;
     int nTasks = xBuckets * yBuckets;
     launch(nTasks,1,1,raytrace_tile_task)
       (width, height, baseWidth, baseHeight,
        raster2camera, camera2world,
        image, id, nodes, triangles);
     hipDeviceSynchronize();
}



extern "C" __host__ void raytrace_ispc_tasks( int width,  int height,
    int baseWidth,  int baseHeight,
    const  float raster2camera[4][4],
    const  float camera2world[4][4],
    float image[],  int id[],
    const  LinearBVHNode nodes[],
    const  Triangle triangles[]) {
  raytrace_ispc_tasks___export<<<1,32>>>( width,  height,
      baseWidth,  baseHeight,
      raster2camera,
      camera2world,
      image,  id,
      nodes,
      triangles);
  hipDeviceSynchronize();
}
