#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
/*
  Copyright (c) 2010-2014, Intel Corporation
  All rights reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are
  met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.

    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

    * Neither the name of Intel Corporation nor the names of its
      contributors may be used to endorse or promote products derived from
      this software without specific prior written permission.


   THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
   IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
   TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
   PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER
   OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
   EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
   PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
   PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
   LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
   NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
   SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "options_defs.h"
#include "cuda_helpers.cuh"

__device__ static inline void __range_reduce_log(float input, float * reduced,
                                      int * exponent) {
    int int_version = __float_as_int(input); //intbits(input);
    // single precision = SEEE EEEE EMMM MMMM MMMM MMMM MMMM MMMM
    // exponent mask    = 0111 1111 1000 0000 0000 0000 0000 0000
    //                    0x7  0xF  0x8  0x0  0x0  0x0  0x0  0x0
    // non-exponent     = 1000 0000 0111 1111 1111 1111 1111 1111
    //                  = 0x8  0x0  0x7  0xF  0xF  0xF  0xF  0xF

    //const int exponent_mask(0x7F800000)
    const int nonexponent_mask = 0x807FFFFF;

    // We want the reduced version to have an exponent of -1 which is -1 + 127 after biasing or 126
    const int exponent_neg1 = (126l << 23);
    // NOTE(boulos): We don't need to mask anything out since we know
    // the sign bit has to be 0. If it's 1, we need to return infinity/nan
    // anyway (log(x), x = +-0 -> infinity, x < 0 -> NaN).
    int biased_exponent = int_version >> 23; // This number is [0, 255] but it means [-127, 128]

    int offset_exponent = biased_exponent + 1; // Treat the number as if it were 2^{e+1} * (1.m)/2
    *exponent = offset_exponent - 127; // get the real value

    // Blend the offset_exponent with the original input (do this in
    // int for now, until I decide if float can have & and &not)
    int blended = (int_version & nonexponent_mask) | (exponent_neg1);
    *reduced = __int_as_float(blended); //floatbits(blended);
}


__device__ static inline float __Logf(const float x_full)
{
#if 1
  return __logf(x_full);
#else
  float reduced;
  int exponent;

  const int NaN_bits = 0x7fc00000;
  const int Neg_Inf_bits = 0xFF800000;
  const float NaN = __int_as_float(NaN_bits); //floatbits(NaN_bits);
  const float neg_inf = __int_as_float(Neg_Inf_bits); //floatbits(Neg_Inf_bits);
  bool use_nan = x_full < 0.f;
  bool use_inf = x_full == 0.f;
  bool exceptional = use_nan || use_inf;
  const float one = 1.0f;

  float patched = exceptional ? one : x_full;
  __range_reduce_log(patched, &reduced, &exponent);

  const float ln2 = 0.693147182464599609375f;

  float x1 = one - reduced;
  const float c1 = 0.50000095367431640625f;
  const float c2 = 0.33326041698455810546875f;
  const float c3 = 0.2519190013408660888671875f;
  const float c4 = 0.17541764676570892333984375f;
  const float c5 = 0.3424419462680816650390625f;
  const float c6 = -0.599632322788238525390625f;
  const float c7 = +1.98442304134368896484375f;
  const float c8 = -2.4899270534515380859375f;
  const float c9 = +1.7491014003753662109375f;

  float result = x1 * c9 + c8;
  result = x1 * result + c7;
  result = x1 * result + c6;
  result = x1 * result + c5;
  result = x1 * result + c4;
  result = x1 * result + c3;
  result = x1 * result + c2;
  result = x1 * result + c1;
  result = x1 * result + one;

  // Equation was for -(ln(red)/(1-red))
  result *= -x1;
  result += (float)(exponent) * ln2;

  return exceptional ? (use_nan ? NaN : neg_inf) : result;
#endif
}

__device__ static inline float __Expf(const float x_full)
{
#if 1
  return __expf(x_full);
#else
  const float ln2_part1 = 0.6931457519f;
  const float ln2_part2 = 1.4286067653e-6f;
  const float one_over_ln2 = 1.44269502162933349609375f;

  float scaled = x_full * one_over_ln2;
  float k_real = floor(scaled);
  int k = (int)k_real;

  // Reduced range version of x
  float x = x_full - k_real * ln2_part1;
  x -= k_real * ln2_part2;

  // These coefficients are for e^x in [0, ln(2)]
  const float one = 1.f;
  const float c2 = 0.4999999105930328369140625f;
  const float c3 = 0.166668415069580078125f;
  const float c4 = 4.16539050638675689697265625e-2f;
  const float c5 = 8.378830738365650177001953125e-3f;
  const float c6 = 1.304379315115511417388916015625e-3f;
  const float c7 = 2.7555381529964506626129150390625e-4f;

  float result = x * c7 + c6;
  result = x * result + c5;
  result = x * result + c4;
  result = x * result + c3;
  result = x * result + c2;
  result = x * result + one;
  result = x * result + one;

  // Compute 2^k (should differ for float and double, but I'll avoid
  // it for now and just do floats)
  const int fpbias = 127;
  int biased_n = k + fpbias;
  bool overflow = k > fpbias;
  // Minimum exponent is -126, so if k is <= -127 (k + 127 <= 0)
  // we've got underflow. -127 * ln(2) -> -88.02. So the most
  // negative float input that doesn't result in zero is like -88.
  bool underflow = (biased_n <= 0);
  const int InfBits = 0x7f800000;
  biased_n <<= 23;
  // Reinterpret this thing as float
  float two_to_the_n = __int_as_float(biased_n); //floatbits(biased_n);
  // Handle both doubles and floats (hopefully eliding the copy for float)
  float elemtype_2n = two_to_the_n;
  result *= elemtype_2n;
//  result = overflow ? floatbits(InfBits) : result;
  result = overflow ? __int_as_float(InfBits) : result;
  result = underflow ? 0.0f : result;
  return result;
#endif
}

// Cumulative normal distribution function
//
__device__
static inline float
CND(float X) {
    float L = fabsf(X);

    float k = 1.0f / (1.0f + 0.2316419f * L);
    float k2 = k*k;
    float k3 = k2*k;
    float k4 = k2*k2;
    float k5 = k3*k2;

    const float invSqrt2Pi = 0.39894228040f;
    float w = (0.31938153f * k - 0.356563782f * k2 + 1.781477937f * k3 +
               -1.821255978f * k4 + 1.330274429f * k5);
    w *= invSqrt2Pi * __Expf(-L * L * .5f);

    if (X > 0.f)
        w = 1.0f - w;
    return w;
}

__global__
void bs_task( float Sa[],  float Xa[],  float Ta[],
    float ra[],  float va[],
    float result[],  int count) {
  if (taskIndex >= taskCount) return;
     int first = taskIndex * (count/taskCount);
     int last = min(count, (int)((taskIndex+1) * (count/taskCount)));

    for (int i = programIndex + first; i < last; i += programCount)
      if (i < last)
    {
        float S = Sa[i], X = Xa[i], T = Ta[i], r = ra[i], v = va[i];

        float d1 = (__Logf(S/X) + (r + v * v * .5f) * T) / (v * sqrtf(T));
        float d2 = d1 - v * sqrtf(T);

        result[i] = S * CND(d1) - X * __Expf(-r * T) * CND(d2);
    }
}

extern "C"
__global__ void
black_scholes_ispc_tasks___export( float Sa[],  float Xa[],  float Ta[],
                          float ra[],  float va[],
                          float result[],  int count) {
  int nTasks = 2048; //count/16384; //max((int)64, (int)count/16384);
  launch(nTasks,1,1,bs_task)
    (Sa, Xa, Ta, ra, va, result, count);
  hipDeviceSynchronize();
}
extern "C"
__host__ void
black_scholes_ispc_tasks( float Sa[],  float Xa[],  float Ta[],
                          float ra[],  float va[],
                          float result[],  int count) {
  black_scholes_ispc_tasks___export<<<1,32>>>(Sa,Xa,Ta,ra,va,result,count);
  hipDeviceSynchronize();
}

/********/


template<int NBEG, int NEND, int STEP>
struct loop
{
  __device__ static void op1(float V[], const float u, const float X, const float S)
  {
    const int j = NBEG;
    float upow = powf(u, (float)(2*j-BINOMIAL_NUM));
    V[j] = max(0.0f, X - S * upow);
    loop<j+STEP,NEND,STEP>::op1(V,u,X,S);
  }
  __device__ static void op2(float V[], const float Pu, const float disc)
  {
    const int j = NBEG;
#pragma unroll
    for ( int k = 0; k < j; ++k)
      V[k] = ((1.0f - Pu) * V[k] + Pu * V[k+ 1]) / disc;
    loop<j+STEP,NEND,STEP>::op2(V, Pu,disc);
  }
};

template<int NEND, int STEP>
struct loop<NEND,NEND,STEP>
{
  __device__ static void op1(float V[], const float u, const float X, const float S) {}
  __device__ static void op2(float V[], const float Pu, const float disc) {}
};

__device__
static inline float
binomial_put(float S, float X, float T, float r, float v)
{

  float V[BINOMIAL_NUM];

  float dt = T / BINOMIAL_NUM;
  float u = exp(v * sqrt(dt));
  float d = 1.f / u;
  float disc = exp(r * dt);
  float Pu = (disc - d) / (u - d);

#if 0  /* slow */
  for ( int j = 0; j < BINOMIAL_NUM; ++j) {
    float upow = powf(u, (float)(2*j-BINOMIAL_NUM));
    V[j] = max(0.0f, X - S * upow);
  }
  for ( int j = BINOMIAL_NUM-1; j >= 0; --j)
    for ( int k = 0; k < j; ++k)
      V[k] = ((1.0f - Pu) * V[k] + Pu * V[k+ 1]) / disc;
#else  /* with loop unrolling, stores resutls in registers */
  loop<0,BINOMIAL_NUM,1>::op1(V,u,X,S);
  loop<BINOMIAL_NUM-1, -1, -1>::op2(V, Pu, disc);
#endif
  return V[0];
}



__global__ void
binomial_task( float Sa[],  float Xa[],
               float Ta[],  float ra[],
               float va[],  float result[],
               int count)
{
  int first = taskIndex * (count/taskCount);
  int last = min(count, (int)((taskIndex+1) * (count/taskCount)));

  for (int i = programIndex + first; i < last; i += programCount)
    if (i < last)
    {
      float S = Sa[i], X = Xa[i], T = Ta[i], r = ra[i], v = va[i];
      result[i] = binomial_put(S, X, T, r, v);
    }
}


extern "C" __global__ void
binomial_put_ispc_tasks___export( float Sa[],  float Xa[],
                         float Ta[],  float ra[],
                         float va[],  float result[],
                         int count) {
  int nTasks = 2048; //count/16384; //max((int)64, (int)count/16384);
  launch(nTasks,1,1,binomial_task)
    (Sa, Xa, Ta, ra, va, result, count);
  hipDeviceSynchronize();
}
extern "C"
__host__ void
binomial_put_ispc_tasks( float Sa[],  float Xa[],  float Ta[],
                          float ra[],  float va[],
                          float result[],  int count) {

  hipDeviceSetCacheConfig (hipFuncCachePreferL1);
  binomial_put_ispc_tasks___export<<<1,32>>>(Sa,Xa,Ta,ra,va,result,count);
  hipDeviceSynchronize();
}
