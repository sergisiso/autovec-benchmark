#include "hip/hip_runtime.h"
/*
  Copyright (c) 2014, Evghenii Gaburov
  All rights reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are
  met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.

    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

    * Neither the name of Intel Corporation nor the names of its
      contributors may be used to endorse or promote products derived from
      this software without specific prior written permission.


   THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
   IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
   TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
   PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER
   OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
   EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
   PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
   PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
   LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
   NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
   SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/*
   Based on mergeSort from CUDA SDK
   */

#include "keyType.h"
#include "cuda_helpers.cuh"
#include <cassert>

#define uniform

#define SAMPLE_STRIDE programCount

#define iDivUp(a,b) (((a) + (b) - 1)/(b))
#define getSampleCount(dividend) (iDivUp((dividend), (SAMPLE_STRIDE)))

#define W (/*sizeof(int)=*/4 * 8)

__device__ static inline
int nextPowerOfTwo(int x)
{
#if 0
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
#else
  return 1U << (W - __clz(x - 1));
#endif
}


__device__ static inline
int binarySearchInclusiveRanks(
    const int val,
    uniform int *data,
    const int L,
    int stride)
{
  if (L == 0)
    return 0;

  int pos = 0;
  for (; stride > 0; stride >>= 1)
  {
    int newPos = min(pos + stride, L);

    if (data[newPos - 1] <= val)
      pos = newPos;
  }

  return pos;
}

__device__ static inline
int binarySearchExclusiveRanks(
    const int val,
    uniform int *data,
    const int L,
    int stride)
{
  if (L == 0)
    return 0;

  int pos = 0;
  for (; stride > 0; stride >>= 1)
  {
    int newPos = min(pos + stride, L);

    if (data[newPos - 1] < val)
      pos = newPos;
  }

  return pos;
}

__device__ static inline
int binarySearchInclusive(
    const Key_t val,
    uniform Key_t *data,
    const int L,
    int stride)
{
  if (L == 0)
    return 0;

  int pos = 0;
  for (; stride > 0; stride >>= 1)
  {
    int newPos = min(pos + stride, L);

    if (data[newPos - 1] <= val)
      pos = newPos;
  }

  return pos;
}

__device__ static inline
int binarySearchExclusive(
    const Key_t val,
    uniform Key_t *data,
    const int L,
    int stride)
{
  if (L == 0)
    return 0;

  int pos = 0;
  for (; stride > 0; stride >>= 1)
  {
    int newPos = min(pos + stride, L);

    if (data[newPos - 1] < val)
      pos = newPos;
  }

  return pos;
}

__device__ static inline
int binarySearchInclusive1(
    const Key_t val,
    Key_t data,
    const uniform int L,
    uniform int stride)
{
  if (L == 0)
    return 0;

  int pos = 0;
  for (; stride > 0; stride >>= 1)
  {
    int newPos = min(pos + stride, L);

    if (shuffle(data,newPos - 1) <= val)
      pos = newPos;
  }

  return pos;
}

__device__ static inline
int binarySearchExclusive1(
    const Key_t val,
    Key_t data,
    const uniform int L,
    uniform int stride)
{
  if (L == 0)
    return 0;

  int pos = 0;
  for (; stride > 0; stride >>= 1)
  {
    int newPos = min(pos + stride, L);

    if (shuffle(data,newPos - 1) < val)
      pos = newPos;
  }

  return pos;
}

////////////////////////////////////////////////////////////////////////////////
// Bottom-level merge sort (binary search-based)
////////////////////////////////////////////////////////////////////////////////
__global__
void mergeSortGangKernel(
    uniform int batchSize,
    uniform Key_t dstKey[],
    uniform Val_t dstVal[],
    uniform Key_t srcKey[],
    uniform Val_t srcVal[])
{
  const uniform int blkIdx = taskIndex;
  const uniform int blkDim = (batchSize + taskCount - 1)/taskCount;
  const uniform int blkBeg =     blkIdx * blkDim;
  const uniform int blkEnd = min(blkBeg + blkDim, batchSize);

  __shared__ Key_t s_key_tmp[2*programCount*4];
  __shared__ Val_t s_val_tmp[2*programCount*4];
  Key_t *s_key = s_key_tmp + warpIdx*(2*programCount);
  Val_t *s_val = s_val_tmp + warpIdx*(2*programCount);

  for (uniform int blk = blkBeg; blk < blkEnd; blk++)
  {
    const uniform int base = blk * (programCount*2);
    s_key[programIndex +            0] = srcKey[base + programIndex +            0];
    s_val[programIndex +            0] = srcVal[base + programIndex +            0];
    s_key[programIndex + programCount] = srcKey[base + programIndex + programCount];
    s_val[programIndex + programCount] = srcVal[base + programIndex + programCount];

    for (uniform int stride = 1; stride < 2*programCount; stride <<= 1)
    {
      const int lPos = programIndex & (stride - 1);
      uniform Key_t *baseKey = s_key + 2 * (programIndex - lPos);
      uniform Val_t *baseVal = s_val + 2 * (programIndex - lPos);

      Key_t keyA = baseKey[lPos +      0];
      Val_t valA = baseVal[lPos +      0];
      Key_t keyB = baseKey[lPos + stride];
      Val_t valB = baseVal[lPos + stride];
      int posA = binarySearchExclusive(keyA, baseKey + stride, stride, stride) + lPos;
      int posB = binarySearchInclusive(keyB, baseKey +      0, stride, stride) + lPos;

      baseKey[posA] = keyA;
      baseVal[posA] = valA;
      baseKey[posB] = keyB;
      baseVal[posB] = valB;
    }

    dstKey[base + programIndex +            0] = s_key[programIndex +            0];
    dstVal[base + programIndex +            0] = s_val[programIndex +            0];
    dstKey[base + programIndex + programCount] = s_key[programIndex + programCount];
    dstVal[base + programIndex + programCount] = s_val[programIndex + programCount];
  }
}

__device__ static inline
void mergeSortGang(
    uniform Key_t dstKey[],
    uniform Val_t dstVal[],
    uniform Key_t srcKey[],
    uniform Val_t srcVal[],
    uniform int batchSize)
{
  uniform int nTasks = batchSize;
  launch (nTasks,1,1,mergeSortGangKernel)(batchSize, dstKey, dstVal, srcKey, srcVal);
  sync;
}

////////////////////////////////////////////////////////////////////////////////
// Merge step 1: generate sample ranks
////////////////////////////////////////////////////////////////////////////////
__global__
void generateSampleRanksKernel(
    uniform int nBlocks,
    uniform int in_ranksA[],
    uniform int in_ranksB[],
    uniform Key_t in_srcKey[],
    uniform int stride,
    uniform int N,
    uniform int totalProgramCount)
{
  const uniform int blkIdx = taskIndex;
  const uniform int blkDim = (nBlocks + taskCount - 1)/taskCount;
  const uniform int blkBeg =     blkIdx * blkDim;
  const uniform int blkEnd = min(blkBeg + blkDim, nBlocks);

  for (uniform int blk = blkBeg; blk < blkEnd; blk++)
  {
    const int pos = blk * programCount + programIndex;
    cif (pos >= totalProgramCount)
      return;

    const int           i = pos & ((stride / SAMPLE_STRIDE) - 1);
    const int segmentBase = (pos - i) * (2 * SAMPLE_STRIDE);

    uniform Key_t * srcKey = in_srcKey + segmentBase;
    uniform int * ranksA = in_ranksA + segmentBase / SAMPLE_STRIDE;
    uniform int * ranksB = in_ranksB + segmentBase / SAMPLE_STRIDE;

    const int segmentElementsA = stride;
    const int segmentElementsB = min(stride, N - segmentBase - stride);
    const int  segmentSamplesA = getSampleCount(segmentElementsA);
    const int  segmentSamplesB = getSampleCount(segmentElementsB);

    if (i < segmentSamplesA)
    {
      ranksA[i] = i * SAMPLE_STRIDE;
      ranksB[i] = binarySearchExclusive(
          srcKey[i * SAMPLE_STRIDE], srcKey + stride,
          segmentElementsB, nextPowerOfTwo(segmentElementsB));
    }

    if (i < segmentSamplesB)
    {
      ranksB[(stride / SAMPLE_STRIDE) + i] = i * SAMPLE_STRIDE;
      ranksA[(stride / SAMPLE_STRIDE) + i] = binarySearchInclusive(
          srcKey[stride + i * SAMPLE_STRIDE], srcKey + 0,
          segmentElementsA, nextPowerOfTwo(segmentElementsA));
    }
  }
}

__device__ static inline
void generateSampleRanks(
    uniform int ranksA[],
    uniform int ranksB[],
    uniform Key_t srcKey[],
    uniform int stride,
    uniform int N)
{
  uniform int lastSegmentElements = N % (2 * stride);
  uniform int threadCount = (lastSegmentElements > stride) ?
    (N + 2 * stride - lastSegmentElements) / (2 * SAMPLE_STRIDE) :
    (N - lastSegmentElements) / (2 * SAMPLE_STRIDE);

  uniform int nBlocks = iDivUp(threadCount, SAMPLE_STRIDE);
  uniform int nTasks = nBlocks;

  launch (nTasks,1,1, generateSampleRanksKernel)(nBlocks, ranksA, ranksB, srcKey, stride, N, threadCount);
  sync;
}
////////////////////////////////////////////////////////////////////////////////
// Merge step 2: generate sample ranks and indices
////////////////////////////////////////////////////////////////////////////////
__global__
void mergeRanksAndIndicesKernel(
    uniform int nBlocks,
    uniform int in_Limits[],
    uniform int in_Ranks[],
    uniform int stride,
    uniform int N,
    uniform int totalProgramCount)
{
  const uniform int blkIdx = taskIndex;
  const uniform int blkDim = (nBlocks + taskCount - 1)/taskCount;
  const uniform int blkBeg =     blkIdx * blkDim;
  const uniform int blkEnd = min(blkBeg + blkDim, nBlocks);

  for (uniform int blk = blkBeg; blk < blkEnd; blk++)
  {
    int pos = blk * programCount + programIndex;
    cif (pos >= totalProgramCount)
      return;

    const int           i = pos & ((stride / SAMPLE_STRIDE) - 1);
    const int segmentBase = (pos - i) * (2 * SAMPLE_STRIDE);
    uniform int *  ranks = in_Ranks  + (pos - i) * 2;
    uniform int * limits = in_Limits + (pos - i) * 2;

    const int segmentElementsA = stride;
    const int segmentElementsB = min(stride, N - segmentBase - stride);
    const int  segmentSamplesA = getSampleCount(segmentElementsA);
    const int  segmentSamplesB = getSampleCount(segmentElementsB);

    if (i < segmentSamplesA)
    {
      int dstPos = binarySearchExclusiveRanks(ranks[i], ranks + segmentSamplesA, segmentSamplesB, nextPowerOfTwo(segmentSamplesB)) + i;
      limits[dstPos] = ranks[i];
    }

    if (i < segmentSamplesB)
    {
      int dstPos = binarySearchInclusiveRanks(ranks[segmentSamplesA + i], ranks, segmentSamplesA, nextPowerOfTwo(segmentSamplesA)) + i;
      limits[dstPos] = ranks[segmentSamplesA + i];
    }
  }
}
__device__ static inline
void mergeRanksAndIndices(
    uniform int limitsA[],
    uniform int limitsB[],
    uniform int ranksA[],
    uniform int ranksB[],
    uniform int stride,
    uniform int N)
{
  const uniform int lastSegmentElements = N % (2 * stride);
  const uniform int threadCount = (lastSegmentElements > stride) ?
    (N + 2 * stride - lastSegmentElements) / (2 * SAMPLE_STRIDE) :
    (N - lastSegmentElements) / (2 * SAMPLE_STRIDE);

  const uniform int nBlocks = iDivUp(threadCount, SAMPLE_STRIDE);
  uniform int nTasks = nBlocks;

  launch (nTasks,1,1,mergeRanksAndIndicesKernel)(
      nBlocks,
      limitsA,
      ranksA,
      stride,
      N,
      threadCount);
  launch (nTasks,1,1, mergeRanksAndIndicesKernel)(
      nBlocks,
      limitsB,
      ranksB,
      stride,
      N,
      threadCount);
  sync;
}


__global__
void mergeElementaryIntervalsKernel(
    uniform int mergePairs,
    uniform Key_t dstKey[],
    uniform Val_t dstVal[],
    uniform Key_t srcKey[],
    uniform Val_t srcVal[],
    uniform int limitsA[],
    uniform int limitsB[],
    uniform int stride,
    uniform int N)
{
  const uniform int blkIdx = taskIndex;
  const uniform int blkDim = (mergePairs + taskCount - 1)/taskCount;
  const uniform int blkBeg =     blkIdx * blkDim;
  const uniform int blkEnd = min(blkBeg + blkDim, mergePairs);

  for (uniform int blk = blkBeg; blk < blkEnd; blk++)
  {
    const int uniform   intervalI =  blk & ((2 * stride) / SAMPLE_STRIDE - 1);
    const int uniform segmentBase = (blk - intervalI) * SAMPLE_STRIDE;

    //Set up threadblk-wide parameters

    const uniform int segmentElementsA = stride;
    const uniform int segmentElementsB = min(stride, N - segmentBase - stride);
    const uniform int  segmentSamplesA = getSampleCount(segmentElementsA);
    const uniform int  segmentSamplesB = getSampleCount(segmentElementsB);
    const uniform int   segmentSamples = segmentSamplesA + segmentSamplesB;

    const uniform int startSrcA = limitsA[blk];
    const uniform int startSrcB = limitsB[blk];
    const uniform int endSrcA   = (intervalI + 1 < segmentSamples) ? limitsA[blk + 1] : segmentElementsA;
    const uniform int endSrcB   = (intervalI + 1 < segmentSamples) ? limitsB[blk + 1] : segmentElementsB;
    const uniform int lenSrcA   = endSrcA - startSrcA;
    const uniform int lenSrcB   = endSrcB - startSrcB;
    const uniform int startDstA = startSrcA + startSrcB;
    const uniform int startDstB = startDstA + lenSrcA;

    //Load main input data

    Key_t keyA, keyB;
    Val_t valA, valB;
    if (programIndex < lenSrcA)
    {
      keyA = srcKey[segmentBase + startSrcA + programIndex];
      valA = srcVal[segmentBase + startSrcA + programIndex];
    }

    if (programIndex < lenSrcB)
    {
      keyB = srcKey[segmentBase + stride + startSrcB + programIndex];
      valB = srcVal[segmentBase + stride + startSrcB + programIndex];
    }

    // Compute destination addresses for merge data
    int dstPosA, dstPosB, dstA = -1, dstB = -1;
    if (any(programIndex < lenSrcA))
      dstPosA = binarySearchExclusive1(keyA, keyB, lenSrcB, SAMPLE_STRIDE) + programIndex;
    if (any(programIndex < lenSrcB))
      dstPosB = binarySearchInclusive1(keyB, keyA, lenSrcA, SAMPLE_STRIDE) + programIndex;

    if (programIndex < lenSrcA && dstPosA < lenSrcA)
      dstA = segmentBase + startDstA + dstPosA;
    dstPosA -= lenSrcA;
    if (programIndex < lenSrcA && dstPosA < lenSrcB)
      dstA = segmentBase + startDstB + dstPosA;

    if (programIndex < lenSrcB && dstPosB < lenSrcA)
      dstB = segmentBase + startDstA + dstPosB;
    dstPosB -= lenSrcA;
    if (programIndex < lenSrcB && dstPosB < lenSrcB)
      dstB = segmentBase + startDstB + dstPosB;

    // store merge data
    if (dstA >= 0)
    {
 //     int dstA = segmentBase + startSrcA + programIndex;
      dstKey[dstA] = keyA;
      dstVal[dstA] = valA;
    }
    if (dstB >= 0)
    {
//      int dstB = segmentBase + stride + startSrcB + programIndex;
      dstKey[dstB] = keyB;
      dstVal[dstB] = valB;
    }
  }

}


__device__ static inline
void mergeElementaryIntervals(
    uniform int nTasks,
    uniform Key_t dstKey[],
    uniform Val_t dstVal[],
    uniform Key_t srcKey[],
    uniform Val_t srcVal[],
    uniform int limitsA[],
    uniform int limitsB[],
    uniform int stride,
    uniform int N)
{
  const uniform int lastSegmentElements = N % (2 * stride);
  const uniform int mergePairs = (lastSegmentElements > stride) ? getSampleCount(N) : (N - lastSegmentElements) / SAMPLE_STRIDE;


  nTasks = mergePairs/(programCount);

  launch (nTasks,1,1, mergeElementaryIntervalsKernel)(
      mergePairs,
      dstKey,
      dstVal,
      srcKey,
      srcVal,
      limitsA,
      limitsB,
      stride,
      N);
  sync;
}

__device__ static uniform int * uniform memPool = NULL;
__device__ static uniform int * uniform ranksA;
__device__ static uniform int * uniform ranksB;
__device__ static uniform int * uniform limitsA;
__device__ static uniform int * uniform limitsB;
__device__ static uniform int nTasks;
__device__ static uniform int MAX_SAMPLE_COUNT = 0;

__global__
void openMergeSort___export()
{
  nTasks = 13*32*13;
  MAX_SAMPLE_COUNT = 8*32 * 131072 / programCount;
  assert(memPool == NULL);
  const uniform int nalloc = MAX_SAMPLE_COUNT * 4;
  memPool = uniform new uniform int[nalloc];
  ranksA  = memPool;
  ranksB  =  ranksA + MAX_SAMPLE_COUNT;
  limitsA =  ranksB + MAX_SAMPLE_COUNT;
  limitsB = limitsA + MAX_SAMPLE_COUNT;
}
extern "C"
void openMergeSort()
{
  openMergeSort___export<<<1,1>>>();
  sync;
}

__global__
void closeMergeSort___export()
{
  assert(memPool != NULL);
  delete memPool;
  memPool = NULL;
}
extern "C"
void closeMergeSort()
{
  closeMergeSort___export<<<1,1>>>();
  sync;
}

__global__
void mergeSort___export(
    uniform Key_t dstKey[],
    uniform Val_t dstVal[],
    uniform Key_t bufKey[],
    uniform Val_t bufVal[],
    uniform Key_t srcKey[],
    uniform Val_t srcVal[],
    uniform int N)
{
  uniform int stageCount = 0;
  for (uniform int stride = 2*programCount; stride < N; stride <<= 1, stageCount++);

  uniform Key_t * uniform iKey, * uniform oKey;
  uniform Val_t * uniform iVal, * uniform oVal;

  if (stageCount & 1)
  {
    iKey = bufKey;
    iVal = bufVal;
    oKey = dstKey;
    oVal = dstVal;
  }
  else
  {
    iKey = dstKey;
    iVal = dstVal;
    oKey = bufKey;
    oVal = bufVal;
  }



  assert(N <= SAMPLE_STRIDE * MAX_SAMPLE_COUNT);
  assert(N % (programCount*2) == 0);

  // k20m: 140 M/s
  {
    // k20m:  2367 M/s
    mergeSortGang(iKey, iVal, srcKey, srcVal, N/(2*programCount));

#if 1
    for (uniform int stride = 2*programCount; stride < N; stride <<= 1)
    {
      const uniform int lastSegmentElements = N % (2 * stride);

      // k20m: 271 M/s
      {
#if 1
        // k20m: 944 M/s
        {
          // k20m:  1396 M/s
          //Find sample ranks and prepare for limiters merge
          generateSampleRanks(ranksA, ranksB, iKey, stride, N);

          // k20m: 2379 M/s
          //Merge ranks and indices
          mergeRanksAndIndices(limitsA, limitsB, ranksA, ranksB, stride, N);
        }
#endif

        // k20m: 371 M/s
        //Merge elementary intervals
        mergeElementaryIntervals(nTasks, oKey, oVal, iKey, iVal, limitsA, limitsB, stride, N);
      }

      if (lastSegmentElements <= stride)
        for (int i = programIndex; i < lastSegmentElements; i += programCount)
          if (i < lastSegmentElements)
          {
            oKey[N-lastSegmentElements+i] = iKey[N-lastSegmentElements+i];
            oVal[N-lastSegmentElements+i] = iVal[N-lastSegmentElements+i];
          }


      {
        uniform Key_t * uniform tmpKey = iKey;
        iKey = oKey;
        oKey = tmpKey;
      }
      {
        uniform Val_t * uniform tmpVal = iVal;
        iVal = oVal;
        oVal = tmpVal;
      }
    }
#endif
  }
}
extern "C"
void mergeSort(
    uniform Key_t dstKey[],
    uniform Val_t dstVal[],
    uniform Key_t bufKey[],
    uniform Val_t bufVal[],
    uniform Key_t srcKey[],
    uniform Val_t srcVal[],
    uniform int N)
{
  mergeSort___export<<<1,32>>>(
      dstKey,
      dstVal,
      bufKey,
      bufVal,
      srcKey,
      srcVal,
      N);
  sync;
}
