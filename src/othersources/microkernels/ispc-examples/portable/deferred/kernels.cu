#include "hip/hip_runtime.h"
/*
  Copyright (c) 2010-2014, Intel Corporation
  All rights reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are
  met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.

    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

    * Neither the name of Intel Corporation nor the names of its
      contributors may be used to endorse or promote products derived from
      this software without specific prior written permission.


   THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
   IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
   TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
   PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER
   OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
   EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
   PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
   PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
   LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
   NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
   SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include "deferred.h"
#include <stdio.h>
#include <assert.h>

#define programCount 32
#define programIndex (threadIdx.x & 31)
#define taskIndex (blockIdx.x*4 + (threadIdx.x >> 5))
#define taskCount (gridDim.x*4)
#define warpIdx (threadIdx.x >> 5)

#define int32 int
#define int16 short
#define int8 char

__device__ static inline float clamp(float v, float low, float high)
{
      return min(max(v, low), high);
}

struct InputDataArrays
{
    float *zBuffer;
    unsigned int16 *normalEncoded_x; // half float
    unsigned int16 *normalEncoded_y; // half float
    unsigned int16 *specularAmount; // half float
    unsigned int16 *specularPower; // half float
    unsigned int8 *albedo_x; // unorm8
    unsigned int8 *albedo_y; // unorm8
    unsigned int8 *albedo_z; // unorm8
    float *lightPositionView_x;
    float *lightPositionView_y;
    float *lightPositionView_z;
    float *lightAttenuationBegin;
    float *lightColor_x;
    float *lightColor_y;
    float *lightColor_z;
    float *lightAttenuationEnd;
};

struct InputHeader
{
    float cameraProj[4][4];
    float cameraNear;
    float cameraFar;

    int32 framebufferWidth;
    int32 framebufferHeight;
    int32 numLights;
    int32 inputDataChunkSize;
    int32 inputDataArrayOffsets[idaNum];
};


///////////////////////////////////////////////////////////////////////////
// Common utility routines

__device__
static inline float
dot3(float x, float y, float z, float a, float b, float c) {
    return (x*a + y*b + z*c);
}


#if 0
static __shared__ int shdata_full[128];
template<typename T, int N>
struct Uniform
{
  T data[(N+programCount-1)/programCount];
  volatile T *shdata;

  __device__ inline Uniform()
  {
    shdata = ((T*)shdata_full) + warpIdx*32;
  }

  __device__ inline int2 get_chunk(const int i) const
  {
    const int elem  = i & (programCount - 1);
    const int chunk = i >> 5;
    shdata[programIndex] = chunk;
    shdata[        elem] = chunk;
    return make_int2(shdata[programIndex], elem);
  }

  __device__ inline const T get(const int i) const
  {
    const int2 idx = get_chunk(i);
    return __shfl(data[idx.x], idx.y);
  }

  __device__ inline void set(const bool active, const int i, T value)
  {
    const int2 idx = get_chunk(i);
    const int chunkIdx = idx.x;
    const int elemIdx = idx.y;
    shdata[programIndex] = data[chunkIdx];
    if (active) shdata[elemIdx] = value;
    data[chunkIdx] = shdata[programIndex];
  }
};
#elif 1
template<typename T, int N>
struct Uniform
{
  union
  {
    T *data;
    int32_t ptr[2];
  };

  __device__ inline Uniform()
  {
    if (programIndex == 0)
      data = (T*)malloc(N*sizeof(T));
    ptr[0] = __shfl(ptr[0], 0);
    ptr[1] = __shfl(ptr[1], 0);
  }
  __device__ inline ~Uniform()
  {
    if (programIndex == 0)
      free(data);
  }

  __device__ inline const T get(const int i) const
  {
    return data[i];
  }

  __device__ inline T* get_ptr(const int i) {return &data[i]; }
  __device__ inline void set(const bool active, const int i, T value)
  {
    if (active)
      data[i] = value;
  }
};

#else
__shared__ int shdata_full[4*MAX_LIGHTS];
template<typename T, int N>
struct Uniform
{
  /* volatile */ T *shdata;

  __device__ Uniform()
  {
    shdata = (T*)&shdata_full[warpIdx*MAX_LIGHTS];
  }

  __device__ inline const T get(const int i) const
  {
    return shdata[i];
  }

  __device__ inline void set(const bool active, const int i, T value)
  {
    if (active)
      shdata[i] = value;
  }
};
#endif


__device__
static inline void
normalize3(float x, float y, float z, float &ox, float &oy, float &oz) {
    float n = rsqrt(x*x + y*y + z*z);
    ox = x * n;
    oy = y * n;
    oz = z * n;
}

__device__ inline
static float reduce_min(float value)
{
#pragma unroll
  for (int i = 4; i >=0; i--)
    value = fminf(value, __shfl_xor(value, 1<<i, 32));
  return value;
}
__device__ inline
static float reduce_max(float value)
{
#pragma unroll
  for (int i = 4; i >=0; i--)
    value = fmaxf(value, __shfl_xor(value, 1<<i, 32));
  return value;
}

#if 0
__device__ inline
static int reduce_sum(int value)
{
#pragma unroll
  for (int i = 4; i >=0; i--)
    value +=  __shfl_xor(value, 1<<i, 32);
  return value;
}
static __device__ __forceinline__ uint shfl_scan_add_step(uint partial, uint up_offset)
{
  uint result;
  asm(
      "{.reg .u32 r0;"
      ".reg .pred p;"
      "shfl.up.b32 r0|p, %1, %2, 0;"
      "@p add.u32 r0, r0, %3;"
      "mov.u32 %0, r0;}"
      : "=r"(result) : "r"(partial), "r"(up_offset), "r"(partial));
  return result;
}
static __device__ __forceinline__ int inclusive_scan_warp(const int value)
{
  uint sum = value;
#pragma unroll
  for(int i = 0; i < 5; ++i)
    sum = shfl_scan_add_step(sum, 1 << i);
  return sum - value;
}
#endif


static __device__ __forceinline__ int lanemask_lt()
{
  int mask;
  asm("mov.u32 %0, %lanemask_lt;" : "=r" (mask));
  return mask;
}
static __device__ __forceinline__ int2 warpBinExclusiveScan(const bool p)
{
  const int b = __ballot(p);
  return make_int2(__popc(b), __popc(b & lanemask_lt()));
}
  __device__ static inline
int packed_store_active(bool active, int* ptr, int value)
{
  const int2 res = warpBinExclusiveScan(active);
  const int idx = res.y;
  const int nactive = res.x;
  if (active)
    ptr[idx] = value;
  return nactive;
}





__device__
static inline float
Unorm8ToFloat32(unsigned int8 u) {
    return (float)u * (1.0f / 255.0f);
}


__device__
static inline unsigned int8
Float32ToUnorm8(float f) {
    return (unsigned int8)(f * 255.0f);
}


__device__
static inline void
ComputeZBounds(
     int32 tileStartX,  int32 tileEndX,
     int32 tileStartY,  int32 tileEndY,
    // G-buffer data
     float zBuffer[],
     int32 gBufferWidth,
    // Camera data
     float cameraProj_33,  float cameraProj_43,
     float cameraNear,  float cameraFar,
    // Output
     float &minZ,
     float &maxZ
    )
{
    // Find Z bounds
    float laneMinZ = cameraFar;
    float laneMaxZ = cameraNear;
    for ( int32 y = tileStartY; y < tileEndY; ++y) {
        for ( int xb = tileStartX; xb < tileEndX; xb += programCount)
        {
          const int x = xb + programIndex;
          if (x >= tileEndX) break;
            // Unproject depth buffer Z value into view space
            float z = zBuffer[y * gBufferWidth + x];
            float viewSpaceZ = cameraProj_43 / (z - cameraProj_33);

            // Work out Z bounds for our samples
            // Avoid considering skybox/background or otherwise invalid pixels
            if ((viewSpaceZ < cameraFar) && (viewSpaceZ >= cameraNear)) {
                laneMinZ = min(laneMinZ, viewSpaceZ);
                laneMaxZ = max(laneMaxZ, viewSpaceZ);
            }
        }
    }
    minZ = reduce_min(laneMinZ);
    maxZ = reduce_max(laneMaxZ);
}


__device__
static inline  int32
IntersectLightsWithTileMinMax(
     int32 tileStartX,  int32 tileEndX,
     int32 tileStartY,  int32 tileEndY,
    // Tile data
     float minZ,
     float maxZ,
    // G-buffer data
     int32 gBufferWidth,  int32 gBufferHeight,
    // Camera data
     float cameraProj_11,  float cameraProj_22,
    // Light Data
     int32 numLights,
     float light_positionView_x_array[],
     float light_positionView_y_array[],
     float light_positionView_z_array[],
     float light_attenuationEnd_array[],
    // Output
     Uniform<int,MAX_LIGHTS> &tileLightIndices
    )
{
     float gBufferScale_x = 0.5f * (float)gBufferWidth;
     float gBufferScale_y = 0.5f * (float)gBufferHeight;

     float frustumPlanes_xy[4] = {
        -(cameraProj_11 * gBufferScale_x),
         (cameraProj_11 * gBufferScale_x),
         (cameraProj_22 * gBufferScale_y),
        -(cameraProj_22 * gBufferScale_y) };
     float frustumPlanes_z[4] = {
         tileEndX - gBufferScale_x,
        -tileStartX + gBufferScale_x,
         tileEndY - gBufferScale_y,
        -tileStartY + gBufferScale_y };

    for ( int i = 0; i < 4; ++i) {
         float norm = rsqrt(frustumPlanes_xy[i] * frustumPlanes_xy[i] +
                                   frustumPlanes_z[i] * frustumPlanes_z[i]);
        frustumPlanes_xy[i] *= norm;
        frustumPlanes_z[i] *= norm;
    }

     int32 tileNumLights = 0;

    for ( int lightIndexB = 0; lightIndexB < numLights; lightIndexB += programCount)
    {
      const int lightIndex = lightIndexB + programIndex;
      if (lightIndex >= numLights) break;

        float light_positionView_z = light_positionView_z_array[lightIndex];
        float light_attenuationEnd = light_attenuationEnd_array[lightIndex];
        float light_attenuationEndNeg = -light_attenuationEnd;

        float d = light_positionView_z - minZ;
        bool inFrustum = (d >= light_attenuationEndNeg);

        d = maxZ - light_positionView_z;
        inFrustum = inFrustum && (d >= light_attenuationEndNeg);

        // This seems better than cif(!inFrustum) ccontinue; here since we
        // don't actually need to mask the rest of this function - this is
        // just a greedy early-out.  Could also structure all of this as
        // nested if() statements, but this a bit easier to read
        if (__ballot(inFrustum) > 0)
        {
            float light_positionView_x = light_positionView_x_array[lightIndex];
            float light_positionView_y = light_positionView_y_array[lightIndex];

            d = light_positionView_z * frustumPlanes_z[0] +
                light_positionView_x * frustumPlanes_xy[0];
            inFrustum = inFrustum && (d >= light_attenuationEndNeg);

            d = light_positionView_z * frustumPlanes_z[1] +
                light_positionView_x * frustumPlanes_xy[1];
            inFrustum = inFrustum && (d >= light_attenuationEndNeg);

            d = light_positionView_z * frustumPlanes_z[2] +
                light_positionView_y * frustumPlanes_xy[2];
            inFrustum = inFrustum && (d >= light_attenuationEndNeg);

            d = light_positionView_z * frustumPlanes_z[3] +
                light_positionView_y * frustumPlanes_xy[3];
            inFrustum = inFrustum && (d >= light_attenuationEndNeg);

            // Pack and store intersecting lights
            const bool active = inFrustum && lightIndex < numLights;
#if 0
            if (__ballot(active) > 0)
              tileNumLights += packed_store_active(active, tileLightIndices.get_ptr(tileNumLights), lightIndex);
#else
            if (__ballot(active) > 0)
            {
              const int2 res = warpBinExclusiveScan(active);
              const int idx = tileNumLights + res.y;
              const int nactive = res.x;
              tileLightIndices.set(active, idx, lightIndex);
              tileNumLights += nactive;
            }
#endif
        }
    }

    return tileNumLights;
}


__device__
static inline   int32
IntersectLightsWithTile(
     int32 tileStartX,  int32 tileEndX,
     int32 tileStartY,  int32 tileEndY,
     int32 gBufferWidth,  int32 gBufferHeight,
    // G-buffer data
     float zBuffer[],
    // Camera data
     float cameraProj_11,  float cameraProj_22,
     float cameraProj_33,  float cameraProj_43,
     float cameraNear,  float cameraFar,
    // Light Data
     int32 numLights,
     float light_positionView_x_array[],
     float light_positionView_y_array[],
     float light_positionView_z_array[],
     float light_attenuationEnd_array[],
    // Output
     Uniform<int,MAX_LIGHTS> &tileLightIndices
    )
{
     float minZ, maxZ;
    ComputeZBounds(tileStartX, tileEndX, tileStartY, tileEndY,
        zBuffer, gBufferWidth, cameraProj_33, cameraProj_43, cameraNear, cameraFar,
        minZ, maxZ);


     int32 tileNumLights = IntersectLightsWithTileMinMax(
        tileStartX, tileEndX, tileStartY, tileEndY, minZ, maxZ,
        gBufferWidth, gBufferHeight, cameraProj_11, cameraProj_22,
        MAX_LIGHTS, light_positionView_x_array, light_positionView_y_array,
        light_positionView_z_array, light_attenuationEnd_array,
        tileLightIndices);

    return tileNumLights;
}


__device__
static inline void
ShadeTile(
     int32 tileStartX,  int32 tileEndX,
     int32 tileStartY,  int32 tileEndY,
     int32 gBufferWidth,  int32 gBufferHeight,
    const  InputDataArrays &inputData,
    // Camera data
     float cameraProj_11,  float cameraProj_22,
     float cameraProj_33,  float cameraProj_43,
    // Light list
     Uniform<int,MAX_LIGHTS> &tileLightIndices,
     int32 tileNumLights,
    // UI
     bool visualizeLightCount,
    // Output
     unsigned int8 framebuffer_r[],
     unsigned int8 framebuffer_g[],
     unsigned int8 framebuffer_b[]
    )
{
    if (tileNumLights == 0 || visualizeLightCount) {
         unsigned int8 c = (unsigned int8)(min(tileNumLights << 2, 255));
        for ( int32 y = tileStartY; y < tileEndY; ++y) {
            for ( int xb = tileStartX ; xb < tileEndX; xb += programCount)
            {
              const int x = xb + programIndex;
              if (x >= tileEndX) continue;
                int32 framebufferIndex = (y * gBufferWidth + x);
                framebuffer_r[framebufferIndex] = c;
                framebuffer_g[framebufferIndex] = c;
                framebuffer_b[framebufferIndex] = c;
            }
        }
    } else {
         float twoOverGBufferWidth = 2.0f / gBufferWidth;
         float twoOverGBufferHeight = 2.0f / gBufferHeight;

        for ( int32 y = tileStartY; y < tileEndY; ++y) {
             float positionScreen_y = -(((0.5f + y) * twoOverGBufferHeight) - 1.f);

            for ( int xb = tileStartX ; xb < tileEndX; xb += programCount)
            {
              const int x = xb + programIndex;
//              if (x >= tileEndX) break;
                int32 gBufferOffset = y * gBufferWidth + x;

                // Reconstruct position and (negative) view vector from G-buffer
                float surface_positionView_x, surface_positionView_y, surface_positionView_z;
                float Vneg_x, Vneg_y, Vneg_z;

                float z = inputData.zBuffer[gBufferOffset];

                // Compute screen/clip-space position
                // NOTE: Mind DX11 viewport transform and pixel center!
                float positionScreen_x = (0.5f + (float)(x)) *
                    twoOverGBufferWidth - 1.0f;

                // Unproject depth buffer Z value into view space
                surface_positionView_z = cameraProj_43 / (z - cameraProj_33);
                surface_positionView_x = positionScreen_x * surface_positionView_z /
                    cameraProj_11;
                surface_positionView_y = positionScreen_y * surface_positionView_z /
                    cameraProj_22;

                // We actually end up with a vector pointing *at* the
                // surface (i.e. the negative view vector)
                normalize3(surface_positionView_x, surface_positionView_y,
                           surface_positionView_z, Vneg_x, Vneg_y, Vneg_z);

                // Reconstruct normal from G-buffer
                float surface_normal_x, surface_normal_y, surface_normal_z;
                asm("// half2float //");
                float normal_x = __half2float(inputData.normalEncoded_x[gBufferOffset]);
                float normal_y = __half2float(inputData.normalEncoded_y[gBufferOffset]);
                asm("// half2float //");

                float f = (normal_x - normal_x * normal_x) + (normal_y - normal_y * normal_y);
                float m = sqrt(4.0f * f - 1.0f);

                surface_normal_x = m * (4.0f * normal_x - 2.0f);
                surface_normal_y = m * (4.0f * normal_y - 2.0f);
                surface_normal_z = 3.0f - 8.0f * f;

                // Load other G-buffer parameters
                float surface_specularAmount =
                    __half2float(inputData.specularAmount[gBufferOffset]);
                float surface_specularPower  =
                    __half2float(inputData.specularPower[gBufferOffset]);
                float surface_albedo_x = Unorm8ToFloat32(inputData.albedo_x[gBufferOffset]);
                float surface_albedo_y = Unorm8ToFloat32(inputData.albedo_y[gBufferOffset]);
                float surface_albedo_z = Unorm8ToFloat32(inputData.albedo_z[gBufferOffset]);

                float lit_x = 0.0f;
                float lit_y = 0.0f;
                float lit_z = 0.0f;
                for ( int32 tileLightIndex = 0; tileLightIndex < tileNumLights;
                     ++tileLightIndex) {
                     int32 lightIndex = tileLightIndices.get(tileLightIndex);

                    // Gather light data relevant to initial culling
                     float light_positionView_x =
                        __ldg(&inputData.lightPositionView_x[lightIndex]);
                     float light_positionView_y =
                        __ldg(&inputData.lightPositionView_y[lightIndex]);
                     float light_positionView_z =
                        __ldg(&inputData.lightPositionView_z[lightIndex]);
                     float light_attenuationEnd =
                        __ldg(&inputData.lightAttenuationEnd[lightIndex]);

                    // Compute light vector
                    float L_x = light_positionView_x - surface_positionView_x;
                    float L_y = light_positionView_y - surface_positionView_y;
                    float L_z = light_positionView_z - surface_positionView_z;

                    float distanceToLight2 = dot3(L_x, L_y, L_z, L_x, L_y, L_z);

                    // Clip at end of attenuation
                    float light_attenutaionEnd2 = light_attenuationEnd * light_attenuationEnd;

                    if (distanceToLight2 < light_attenutaionEnd2) {
                        float distanceToLight = sqrt(distanceToLight2);

                        // HLSL "rcp" is allowed to be fairly inaccurate
                        float distanceToLightRcp = 1.0f/distanceToLight;
                        L_x *= distanceToLightRcp;
                        L_y *= distanceToLightRcp;
                        L_z *= distanceToLightRcp;

                        // Start computing brdf
                        float NdotL = dot3(surface_normal_x, surface_normal_y,
                                           surface_normal_z, L_x, L_y, L_z);

                        // Clip back facing
                        if (NdotL > 0.0f) {
                             float light_attenuationBegin =
                                inputData.lightAttenuationBegin[lightIndex];

                            // Light distance attenuation (linstep)
                            float lightRange = (light_attenuationEnd - light_attenuationBegin);
                            float falloffPosition = (light_attenuationEnd - distanceToLight);
                            float attenuation = min(falloffPosition / lightRange, 1.0f);

                            float H_x = (L_x - Vneg_x);
                            float H_y = (L_y - Vneg_y);
                            float H_z = (L_z - Vneg_z);
                            normalize3(H_x, H_y, H_z, H_x, H_y, H_z);

                            float NdotH = dot3(surface_normal_x, surface_normal_y,
                                               surface_normal_z, H_x, H_y, H_z);
                            NdotH = max(NdotH, 0.0f);

                            float specular = pow(NdotH, surface_specularPower);
                            float specularNorm = (surface_specularPower + 2.0f) *
                                (1.0f / 8.0f);
                            float specularContrib = surface_specularAmount *
                                specularNorm * specular;

                            float k = attenuation * NdotL * (1.0f + specularContrib);

                             float light_color_x = inputData.lightColor_x[lightIndex];
                             float light_color_y = inputData.lightColor_y[lightIndex];
                             float light_color_z = inputData.lightColor_z[lightIndex];

                            float lightContrib_x = surface_albedo_x * light_color_x;
                            float lightContrib_y = surface_albedo_y * light_color_y;
                            float lightContrib_z = surface_albedo_z * light_color_z;

                            lit_x += lightContrib_x * k;
                            lit_y += lightContrib_y * k;
                            lit_z += lightContrib_z * k;
                        }
                    }
                }

                // Gamma correct
                // These pows are pretty slow right now, but we can do
                // something faster if really necessary to squeeze every
                // last bit of performance out of it
                float gamma = 1.0 / 2.2f;
                lit_x = pow(clamp(lit_x, 0.0f, 1.0f), gamma);
                lit_y = pow(clamp(lit_y, 0.0f, 1.0f), gamma);
                lit_z = pow(clamp(lit_z, 0.0f, 1.0f), gamma);

                framebuffer_r[gBufferOffset] = Float32ToUnorm8(lit_x);
                framebuffer_g[gBufferOffset] = Float32ToUnorm8(lit_y);
                framebuffer_b[gBufferOffset] = Float32ToUnorm8(lit_z);
            }
        }
    }
}


///////////////////////////////////////////////////////////////////////////
// Static decomposition

__global__ void
RenderTile( int num_groups_x,  int num_groups_y,
           const  InputHeader *inputHeaderPtr,
           const  InputDataArrays *inputDataPtr,
            int visualizeLightCount,
           // Output
            unsigned int8 framebuffer_r[],
            unsigned int8 framebuffer_g[],
            unsigned int8 framebuffer_b[]) {
  if (taskIndex >= taskCount) return;

  const  InputHeader inputHeader = *inputHeaderPtr;
  const  InputDataArrays inputData = *inputDataPtr;
     int32 group_y = taskIndex / num_groups_x;
     int32 group_x = taskIndex % num_groups_x;

     int32 tile_start_x = group_x * MIN_TILE_WIDTH;
     int32 tile_start_y = group_y * MIN_TILE_HEIGHT;
     int32 tile_end_x = tile_start_x + MIN_TILE_WIDTH;
     int32 tile_end_y = tile_start_y + MIN_TILE_HEIGHT;

     int framebufferWidth = inputHeader.framebufferWidth;
     int framebufferHeight = inputHeader.framebufferHeight;
     float cameraProj_00 = inputHeader.cameraProj[0][0];
     float cameraProj_11 = inputHeader.cameraProj[1][1];
     float cameraProj_22 = inputHeader.cameraProj[2][2];
     float cameraProj_32 = inputHeader.cameraProj[3][2];

    // Light intersection: figure out which lights illuminate this tile.
     Uniform<int,MAX_LIGHTS> tileLightIndices;  // Light list for the tile
#if 1
     int numTileLights =
        IntersectLightsWithTile(tile_start_x, tile_end_x,
                                tile_start_y, tile_end_y,
                                framebufferWidth, framebufferHeight,
                                inputData.zBuffer,
                                cameraProj_00, cameraProj_11,
                                cameraProj_22, cameraProj_32,
                                inputHeader.cameraNear, inputHeader.cameraFar,
                                MAX_LIGHTS,
                                inputData.lightPositionView_x,
                                inputData.lightPositionView_y,
                                inputData.lightPositionView_z,
                                inputData.lightAttenuationEnd,
                                tileLightIndices);

    // And now shade the tile, using the lights in tileLightIndices
    ShadeTile(tile_start_x, tile_end_x, tile_start_y, tile_end_y,
              framebufferWidth, framebufferHeight, inputData,
              cameraProj_00, cameraProj_11, cameraProj_22, cameraProj_32,
              tileLightIndices, numTileLights, visualizeLightCount,
              framebuffer_r, framebuffer_g, framebuffer_b);
#endif
}


extern "C" __global__ void
RenderStatic___export( InputHeader inputHeaderPtr[],
              InputDataArrays inputDataPtr[],
              int visualizeLightCount,
             // Output
              unsigned int8 framebuffer_r[],
              unsigned int8 framebuffer_g[],
              unsigned int8 framebuffer_b[]) {

  const  InputHeader inputHeader = *inputHeaderPtr;
  const  InputDataArrays inputData = *inputDataPtr;


     int num_groups_x = (inputHeader.framebufferWidth +
                                MIN_TILE_WIDTH - 1) / MIN_TILE_WIDTH;
     int num_groups_y = (inputHeader.framebufferHeight +
                                MIN_TILE_HEIGHT - 1) / MIN_TILE_HEIGHT;
     int num_groups = num_groups_x * num_groups_y;

    // Launch a task to render each tile, each of which is MIN_TILE_WIDTH
    // by MIN_TILE_HEIGHT pixels.
     if (programIndex == 0)
       RenderTile<<<(num_groups+4-1)/4,128>>>(num_groups_x, num_groups_y,
           inputHeaderPtr, inputDataPtr, visualizeLightCount,
           framebuffer_r, framebuffer_g, framebuffer_b);
     hipDeviceSynchronize();
}
extern "C" __host__ void
RenderStatic( InputHeader inputHeaderPtr[],
              InputDataArrays inputDataPtr[],
              int visualizeLightCount,
             // Output
              unsigned int8 framebuffer_r[],
              unsigned int8 framebuffer_g[],
              unsigned int8 framebuffer_b[]) {
  RenderStatic___export<<<1,32>>>( inputHeaderPtr,
              inputDataPtr,
              visualizeLightCount,
             // Output
              framebuffer_r,
              framebuffer_g,
              framebuffer_b);
     hipDeviceSynchronize();
}
