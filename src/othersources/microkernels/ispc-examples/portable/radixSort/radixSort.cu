#include "hip/hip_runtime.h"
/*
  Copyright (c) 2014, Evghenii Gaburov
  All rights reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are
  met:

    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.

    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

    * Neither the name of Intel Corporation nor the names of its
      contributors may be used to endorse or promote products derived from
      this software without specific prior written permission.


   THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
   IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
   TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
   PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER
   OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
   EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
   PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
   PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
   LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
   NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
   SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/*
   Based on radixSort from  http://www.moderngpu.com
   */

#include "cuda_helpers.cuh"
#include <cassert>

#define NUMBITS 8
#define NUMDIGITS (1<<NUMBITS)

typedef long long Key;

__forceinline__ __device__ int atomic_add_global(int* ptr, int value)
{
  return atomicAdd(ptr, value);
}

static __device__ __forceinline__ int shfl_scan_add_step(int partial, int up_offset)
{
  int result;
  asm(
      "{.reg .u32 r0;"
      ".reg .pred p;"
      "shfl.up.b32 r0|p, %1, %2, 0;"
      "@p add.u32 r0, r0, %3;"
      "mov.u32 %0, r0;}"
      : "=r"(result) : "r"(partial), "r"(up_offset), "r"(partial));
  return result;
}

__forceinline__ __device__ int exclusive_scan_add(int value)
{
  int mysum = value;
#pragma unroll
  for(int i = 0; i < 5; ++i)
    mysum = shfl_scan_add_step(mysum, 1 << i);
  return mysum - value;
}

__global__
void countPass(
    const  Key keysAll[],
    Key sortedAll[],
    const  int bit,
    const  int numElements,
    int countsAll[],
    int countsGlobal[])
{
  const  int  blkIdx = taskIndex;
  const  int numBlocks = taskCount;
  const  int  blkDim = (numElements + numBlocks - 1) / numBlocks;

  const  int mask = (1 << NUMBITS) - 1;

  const  Key *  keys   =   keysAll + blkIdx*blkDim;
  Key *  sorted = sortedAll + blkIdx*blkDim;
  int *      counts = countsAll + blkIdx*NUMDIGITS;
  const  int           nloc = min(numElements - blkIdx*blkDim, blkDim);

#pragma unroll 8
  for (int digit = programIndex; digit < NUMDIGITS; digit += programCount)
    counts[digit] = 0;

  for (int i = programIndex; i < nloc; i += programCount)
    if (i < nloc)
    {
      sorted[i] = keys[i];
      const int key = mask & ((unsigned int)keys[i] >> bit);
      atomic_add_global(&counts[key], 1);
    }

#pragma unroll 8
  for (int digit = programIndex; digit < NUMDIGITS; digit += programCount)
    atomic_add_global(&countsGlobal[digit], counts[digit]);
}

__global__
void sortPass(
    Key keysAll[],
    Key sorted[],
    int bit,
    int numElements,
    int digitOffsetsAll[])
{
  const  int  blkIdx = taskIndex;
  const  int numBlocks = taskCount;

  const  int  blkDim = (numElements + numBlocks - 1) / numBlocks;


  const  int keyIndex = blkIdx * blkDim;
  Key *  keys = keysAll + keyIndex;


  const  int nloc = min(numElements - keyIndex, blkDim);

  const  int mask = (1 << NUMBITS) - 1;

  /* copy digit offset from Gmem to Lmem */
#if 1
  __shared__ int digitOffsets_sh[NUMDIGITS*4];
  volatile int *digitOffsets = digitOffsets_sh + warpIdx*NUMDIGITS;
  for (int digit = programIndex; digit < NUMDIGITS; digit += programCount)
    digitOffsets[digit] = digitOffsetsAll[blkIdx*NUMDIGITS + digit];
#else
  int *digitOffsets = &digitOffsetsAll[blkIdx*NUMDIGITS];
#endif


  for (int i = programIndex; i < nloc; i += programCount)
    if (i < nloc)
    {
      const int key = mask & ((unsigned int)keys[i] >> bit);
      int scatter;
      /* not a vector friendly loop */
#pragma unroll 1  /* needed, otherwise compiler unroll and optimizes the result :S */
      for (int iv = 0; iv < programCount; iv++)
        if (programIndex == iv)
          scatter = digitOffsets[key]++;
      sorted [scatter] = keys[i];
    }
}

__global__
void partialScanLocal(
    int numBlocks,
    int excScanAll[],
    int  countsAll[],
    int partialSumAll[])
{
  const  int  blkIdx = taskIndex;

  const  int  blkDim = (numBlocks+taskCount-1)/taskCount;
  const  int      bbeg = blkIdx * blkDim;
  const  int      bend = min(bbeg + blkDim, numBlocks);

  int (*   countsBlock)[NUMDIGITS] = ( int (*)[NUMDIGITS])countsAll;
  int (*  excScanBlock)[NUMDIGITS] = ( int (*)[NUMDIGITS])excScanAll;
  int (*    partialSum)[NUMDIGITS] = ( int (*)[NUMDIGITS])partialSumAll;

#pragma unroll 8
  for (int digit = programIndex; digit < NUMDIGITS; digit += programCount)
  {
    int prev = bbeg == 0 ? excScanBlock[0][digit] : 0;
    for ( int block = bbeg; block < bend; block++)
    {
      const int y = countsBlock[block][digit];
      excScanBlock[block][digit] = prev;
      prev += y;
    }
    partialSum[blkIdx][digit] = excScanBlock[bend-1][digit] + countsBlock[bend-1][digit];
  }
}

__global__
void partialScanGlobal(
    const  int numBlocks,
    int partialSumAll[],
    int prefixSumAll[])
{
  int (*  partialSum)[NUMDIGITS] = ( int (*)[NUMDIGITS])partialSumAll;
  int (*   prefixSum)[NUMDIGITS] = ( int (*)[NUMDIGITS]) prefixSumAll;
  const  int digit = taskIndex;
  int carry = 0;
  for (int block = programIndex;  block < numBlocks; block += programCount)
  {
    const int value = partialSum[block][digit];
    const int scan  = exclusive_scan_add(value);
    if (block < numBlocks)
      prefixSum[block][digit] = scan + carry;
    carry += __shfl(scan+value, programCount-1);
  }
}

__global__
void completeScanGlobal(
    int numBlocks,
    int excScanAll[],
    int carryValueAll[])
{
  const  int  blkIdx = taskIndex;
  const  int  blkDim = (numBlocks+taskCount-1)/taskCount;
  const  int      bbeg = blkIdx * blkDim;
  const  int      bend = min(bbeg  + blkDim, numBlocks);

  int (*  excScanBlock)[NUMDIGITS] = ( int (*)[NUMDIGITS])excScanAll;
  int (*    carryValue)[NUMDIGITS] = ( int (*)[NUMDIGITS])carryValueAll;

#pragma unroll 8
  for (int digit = programIndex; digit < NUMDIGITS; digit += programCount)
  {
    const int carry = carryValue[blkIdx][digit];
    for ( int block = bbeg; block < bend; block++)
      excScanBlock[block][digit] += carry;
  }
}

__device__ static
inline void radixExclusiveScan(
    const  int numBlocks,
    int excScanPtr[],
    int  countsPtr[],
    int partialSum[],
    int  prefixSum[])
{
  const  int scale = 8;
  launch (numBlocks/scale, 1,1, partialScanLocal)(numBlocks, excScanPtr, countsPtr, partialSum);
  sync;

  launch (NUMDIGITS,1,1,partialScanGlobal) (numBlocks/scale, partialSum, prefixSum);
  sync;

  launch (numBlocks/scale,1,1, completeScanGlobal) (numBlocks, excScanPtr, prefixSum);
  sync;
}

__device__ static  int *  memoryPool = NULL;
__device__ static  int numBlocks;
__device__ static  int nSharedCounts;
__device__ static  int nCountsGlobal;
__device__ static  int nExcScan;
__device__ static  int nCountsBlock;
__device__ static  int nPartialSum;
__device__ static  int nPrefixSum;

__device__ static  int *  sharedCounts;
__device__ static  int *  countsGlobal;
__device__ static  int *  excScan;
__device__ static  int *  counts;
__device__ static  int *  partialSum;
__device__ static  int *  prefixSum;

__device__ static  int numElementsBuf = 0;
__device__ static  Key *  bufKeys;

__global__
void radixSort_alloc___export(const  int n)
{
  assert(memoryPool == NULL);
  numBlocks     = 13*32*4;
  nSharedCounts = NUMDIGITS*numBlocks;
  nCountsGlobal = NUMDIGITS;
  nExcScan      = NUMDIGITS*numBlocks;
  nCountsBlock  = NUMDIGITS*numBlocks;
  nPartialSum   = NUMDIGITS*numBlocks;
  nPrefixSum    = NUMDIGITS*numBlocks;


  const  int nalloc =
    nSharedCounts +
    nCountsGlobal +
    nExcScan +
    nCountsBlock +
    nPartialSum +
    nPrefixSum;

  if (programIndex == 0)
    memoryPool =  new  int[nalloc];

  sharedCounts = memoryPool;
  countsGlobal = sharedCounts + nSharedCounts;
  excScan      = countsGlobal + nCountsGlobal;
  counts       = excScan      + nExcScan;
  partialSum   = counts       + nCountsBlock;
  prefixSum    = partialSum   + nPartialSum;
}

extern "C"
void radixSort_alloc(const  int n)
{
  radixSort_alloc___export<<<1,32>>>(n);
  sync;
}


__device__  static
void radixSort_freeBufKeys()
{
  if (numElementsBuf > 0)
  {
    if (programIndex == 0)
      delete bufKeys;
    numElementsBuf = 0;
  }
}

__global__ void radixSort_free___export()
{
  assert(memoryPool != NULL);
  if (programIndex == 0)
    delete memoryPool;
  memoryPool = NULL;

  radixSort_freeBufKeys();
}
extern "C"
void radixSort_free()
{
  radixSort_free___export<<<1,32>>>();
  sync;
}

__global__ void radixSort___export(
    const  int numElements,
    Key keys[],
    const  int nBits)
{
#ifdef __NVPTX__
  assert((numBlocks & 3) == 0);  /* task granularity on Kepler is 4 */
#endif

  if (numElementsBuf < numElements)
    radixSort_freeBufKeys();
  if (numElementsBuf == 0)
  {
    numElementsBuf = numElements;
    if (programIndex == 0)
      bufKeys =  new  Key[numElementsBuf];
  }

  const  int blkDim  = (numElements + numBlocks - 1) / numBlocks;

  for ( int bit = 0; bit < nBits; bit += NUMBITS)
  {
    /* initialize histogram for each digit */
    for (int digit = programIndex; digit < NUMDIGITS; digit += programCount)
      countsGlobal[digit] = 0;

    /* compute histogram for each digit */
    launch (numBlocks,1,1, countPass)(keys, bufKeys, bit, numElements, counts, countsGlobal);
    sync;

    /* exclusive scan on global histogram */
    int carry = 0;
    excScan[0] = 0;
#pragma unroll 8
    for (int digit = programIndex; digit < NUMDIGITS; digit += programCount)
    {
      const int value = countsGlobal[digit];
      const int scan  = exclusive_scan_add(value);
      excScan[digit] = scan + carry;
      carry += __shfl(scan+value, programCount-1);
    }

    /* computing offsets for each digit */
    radixExclusiveScan(numBlocks, excScan, counts, partialSum, prefixSum);

    /* sorting */
    launch (numBlocks,1,1,
      sortPass)(
          bufKeys,
          keys,
          bit,
          numElements,
          excScan);
    sync;
  }
}

extern "C"
void radixSort(
    const  int numElements,
    Key keys[],
    const  int nBits)
{
  hipDeviceSetCacheConfig ( hipFuncCachePreferEqual );
  radixSort___export<<<1,32>>>(numElements, keys, nBits);
  sync;
}
