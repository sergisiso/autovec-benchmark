#include "hip/hip_runtime.h"
// -*- mode: c++ -*-
/*
   Copyright (c) 2010-2014, Intel Corporation
   All rights reserved.

   Redistribution and use in source and binary forms, with or without
   modification, are permitted provided that the following conditions are
met:

 * Redistributions of source code must retain the above copyright
 notice, this list of conditions and the following disclaimer.

 * Redistributions in binary form must reproduce the above copyright
 notice, this list of conditions and the following disclaimer in the
 documentation and/or other materials provided with the distribution.

 * Neither the name of Intel Corporation nor the names of its
 contributors may be used to endorse or promote products derived from
 this software without specific prior written permission.


 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
 IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
 TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER
 OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
 LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
/*
   Based on Syoyo Fujita's aobench: http://code.google.com/p/aobench
   */

#include "cuda_helpers.cuh"

#define NAO_SAMPLES        8
//#define M_PI 3.1415926535f

#define vec Float3
struct Float3
{
  float x,y,z;

  __device__ friend Float3 operator+(const Float3 a, const Float3 b)
  {
    Float3 c;
    c.x = a.x+b.x;
    c.y = a.y+b.y;
    c.z = a.z+b.z;
    return c;
  }
  __device__ friend Float3 operator-(const Float3 a, const Float3 b)
  {
    Float3 c;
    c.x = a.x-b.x;
    c.y = a.y-b.y;
    c.z = a.z-b.z;
    return c;
  }
  __device__ friend Float3 operator/(const Float3 a, const Float3 b)
  {
    Float3 c;
    c.x = a.x/b.x;
    c.y = a.y/b.y;
    c.z = a.z/b.z;
    return c;
  }
  __device__ friend Float3 operator/(const float a, const Float3 b)
  {
    Float3 c;
    c.x = a/b.x;
    c.y = a/b.y;
    c.z = a/b.z;
    return c;
  }
  __device__ friend Float3 operator*(const Float3 a, const Float3 b)
  {
    Float3 c;
    c.x = a.x*b.x;
    c.y = a.y*b.y;
    c.z = a.z*b.z;
    return c;
  }
  __device__ friend Float3 operator*(const Float3 a, const float b)
  {
    Float3 c;
    c.x = a.x*b;
    c.y = a.y*b;
    c.z = a.z*b;
    return c;
  }
};

///////////////////////////////////////////////////////////////////////////
// RNG stuff

struct RNGState {
    unsigned int z1, z2, z3, z4;
};

__device__
static inline unsigned int random(RNGState * state)
{
    unsigned int b;

    b  = ((state->z1 << 6) ^ state->z1) >> 13;
    state->z1 = ((state->z1 & 4294967294U) << 18) ^ b;
    b  = ((state->z2 << 2) ^ state->z2) >> 27;
    state->z2 = ((state->z2 & 4294967288U) << 2) ^ b;
    b  = ((state->z3 << 13) ^ state->z3) >> 21;
    state->z3 = ((state->z3 & 4294967280U) << 7) ^ b;
    b  = ((state->z4 << 3) ^ state->z4) >> 12;
    state->z4 = ((state->z4 & 4294967168U) << 13) ^ b;
    return (state->z1 ^ state->z2 ^ state->z3 ^ state->z4);
}


__device__
static inline float frandom(RNGState * state)
{
    unsigned int irand = random(state);
    irand &= (1ul<<23)-1;
    return __int_as_float(0x3F800000 | irand)-1.0f;
}

__device__
static inline void seed_rng(RNGState * state,
                            unsigned int seed) {
    state->z1 = seed;
    state->z2 = seed ^ 0xbeeff00d;
    state->z3 = ((seed & 0xfffful) << 16) | (seed >> 16);
    state->z4 = (((seed & 0xfful) << 24) | ((seed & 0xff00ul)  << 8) |
                 ((seed & 0xff0000ul) >> 8) | (seed & 0xff000000ul) >> 24);
}



struct Isect {
  float      t;
  vec        p;
  vec        n;
  int        hit;
};

struct Sphere {
  vec        center;
  float      radius;
};

struct Plane {
  vec    p;
  vec    n;
};

struct Ray {
  vec org;
  vec dir;
};

__device__
static inline float dot(vec a, vec b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__
static inline vec vcross(vec v0, vec v1) {
  vec ret;
  ret.x = v0.y * v1.z - v0.z * v1.y;
  ret.y = v0.z * v1.x - v0.x * v1.z;
  ret.z = v0.x * v1.y - v0.y * v1.x;
  return ret;
}

__device__
static inline void vnormalize(vec &v) {
  float len2 = dot(v, v);
  float invlen = rsqrt(len2);
  v = v*invlen;
}


__device__
static inline void
ray_plane_intersect(Isect &isect,const  Ray &ray, const  Plane &plane) {
  float d = -dot(plane.p, plane.n);
  float v = dot(ray.dir, plane.n);

#if 0
  if (abs(v) < 1.0f-17)
    return;
  else {
    float t = -(dot(ray.org, plane.n) + d) / v;

    if ((t > 0.0) && (t < isect.t)) {
      isect.t = t;
      isect.hit = 1;
      isect.p = ray.org + ray.dir * t;
      isect.n = plane.n;
    }
  }
#else
    if (abs(v) <= 1.0e-17)
      return;
    float t = -(dot(ray.org, plane.n) + d) / v;
    if ((t > 0.0) && (t < isect.t)) {
      isect.t = t;
      isect.hit = 1;
      isect.p = ray.org + ray.dir * t;
      isect.n = plane.n;
    }
#endif
}


__device__
static inline void
ray_sphere_intersect(Isect &isect,const  Ray &ray, const Sphere &sphere) {
  vec rs = ray.org - sphere.center;

  float B = dot(rs, ray.dir);
  float C = dot(rs, rs) - sphere.radius * sphere.radius;
  float D = B * B - C;

#if 0
  if (D > 0.) {
    float t = -B - sqrt(D);

    if ((t > 0.0) && (t < isect.t)) {
      isect.t = t;
      isect.hit = 1;
      isect.p = ray.org +  ray.dir * t;
      isect.n = isect.p - sphere.center;
      vnormalize(isect.n);
    }
  }
#else
    if (D <= 0.0f)
      return;

    float t = -B - sqrt(D);

    if ((t > 0.0) && (t < isect.t)) {
      isect.t = t;
      isect.hit = 1;
      isect.p = ray.org +  ray.dir * t;
      isect.n = isect.p - sphere.center;
      vnormalize(isect.n);
    }
#endif

}


__device__
static inline void
orthoBasis(vec basis[3], vec n) {
  basis[2] = n;
  basis[1].x = 0.0f; basis[1].y = 0.0f; basis[1].z = 0.0f;

  if ((n.x < 0.6f) && (n.x > -0.6f)) {
    basis[1].x = 1.0f;
  } else if ((n.y < 0.6f) && (n.y > -0.6f)) {
    basis[1].y = 1.0f;
  } else if ((n.z < 0.6f) && (n.z > -0.6f)) {
    basis[1].z = 1.0f;
  } else {
    basis[1].x = 1.0f;
  }

  basis[0] = vcross(basis[1], basis[2]);
  vnormalize(basis[0]);

  basis[1] = vcross(basis[2], basis[0]);
  vnormalize(basis[1]);
}


__device__
static inline float
ambient_occlusion(Isect &isect,  const Plane &plane, const  Sphere spheres[3],
    RNGState &rngstate) {
  float eps = 0.0001f;
  vec p; //, n;
  vec basis[3];
  float occlusion = 0.0f;

  p = isect.p + isect.n * eps;

  orthoBasis(basis, isect.n);

  const  int ntheta = NAO_SAMPLES;
  const  int nphi   = NAO_SAMPLES;
  for ( int j = 0; j < ntheta; j++) {
    for ( int i = 0; i < nphi; i++) {
      Ray ray;
      Isect occIsect;

      float theta = sqrt(frandom(&rngstate));
      float phi   = 2.0f * M_PI * frandom(&rngstate);
      float x = cos(phi) * theta;
      float y = sin(phi) * theta;
      float z = sqrtf(1.0f - theta * theta);

      // local . global
      float rx = x * basis[0].x + y * basis[1].x + z * basis[2].x;
      float ry = x * basis[0].y + y * basis[1].y + z * basis[2].y;
      float rz = x * basis[0].z + y * basis[1].z + z * basis[2].z;

      ray.org = p;
      ray.dir.x = rx;
      ray.dir.y = ry;
      ray.dir.z = rz;

      occIsect.t   = 1.0f+17;
      occIsect.hit = 0;

      for ( int snum = 0; snum < 3; ++snum)
        ray_sphere_intersect(occIsect, ray, spheres[snum]);
      ray_plane_intersect (occIsect, ray, plane);

      if (occIsect.hit) occlusion += 1.0f;
    }
  }

  occlusion = (ntheta * nphi - occlusion) / (float)(ntheta * nphi);
  return occlusion;
}


/* Compute the image for the scanlines from [y0,y1), for an overall image
   of width w and height h.
   */
__device__
static inline void ao_tiles(
     int x0,  int x1,
     int y0,  int y1,
     int w,  int h,
     int nsubsamples,
     float image[])
{
  const  Plane plane = { { 0.0f, -0.5f, 0.0f }, { 0.f, 1.f, 0.f } };
  const  Sphere spheres[3] = {
    { { -2.0f, 0.0f, -3.5f }, 0.5f },
    { { -0.5f, 0.0f, -3.0f }, 0.5f },
    { { 1.0f, 0.0f, -2.2f }, 0.5f } };
  RNGState rngstate;

  seed_rng(&rngstate, programIndex + (y0 << (programIndex & 15)));
  float invSamples = 1.f / nsubsamples;
  for ( int y = y0; y < y1; y++)
    for ( int x = programIndex+x0; x < x1; x += programCount)
    {
      const int offset = 3 * (y * w + x);
      float res = 0.0f;

      for ( int u = 0; u < nsubsamples; u++)
        for ( int v = 0; v < nsubsamples; v++)
        {
          float du = (float)u * invSamples, dv = (float)v * invSamples;

          // Figure out x,y pixel in NDC
          float px =  (x + du - (w / 2.0f)) / (w / 2.0f);
          float py = -(y + dv - (h / 2.0f)) / (h / 2.0f);
          float ret = 0.f;
          Ray ray;
          Isect isect;

          ray.org.x = 0.0f;
          ray.org.y = 0.0f;
          ray.org.z = 0.0f;

          // Poor man's perspective projection
          ray.dir.x = px;
          ray.dir.y = py;
          ray.dir.z = -1.0;
          vnormalize(ray.dir);

          isect.t   = 1.0e+17;
          isect.hit = 0;

          for ( int snum = 0; snum < 3; ++snum)
            ray_sphere_intersect(isect, ray, spheres[snum]);
          ray_plane_intersect(isect, ray, plane);

          // Note use of 'coherent' if statement; the set of rays we
          // trace will often all hit or all miss the scene
          if (any(isect.hit)) {
            ret = isect.hit*ambient_occlusion(isect, plane, spheres, rngstate);
            ret *= invSamples * invSamples;
            res += ret;
          }
        }

      if (x < x1)
      {
        image[offset  ] = res;
        image[offset+1] = res;
        image[offset+2] = res;
      }
    }
}



#define TILEX 64
#define TILEY 4

extern "C"
__global__
void ao_task( int width,  int height,
     int nsubsamples,  float image[])
{
  if (taskIndex0 >= taskCount0) return;
  if (taskIndex1 >= taskCount1) return;

  const  int x0 = taskIndex0 * TILEX;
  const  int x1 = min(x0 + TILEX, width);

  const  int y0 = taskIndex1 * TILEY;
  const  int y1 = min(y0 + TILEY, height);
  ao_tiles(x0,x1,y0,y1, width, height, nsubsamples, image);
}

extern "C"
__global__
void ao_ispc_tasks___export(
    int w, int h, int nsubsamples,
    float image[])
{
  const int ntilex = (w+TILEX-1)/TILEX;
  const int ntiley = (h+TILEY-1)/TILEY;
  launch(ntilex,ntiley,1,ao_task)(w,h,nsubsamples,image);
  hipDeviceSynchronize();
}

extern "C"
__host__ void ao_ispc_tasks(
    int w, int h, int nsubsamples,
    float image[])
{
  ao_ispc_tasks___export<<<1,32>>>(w,h,nsubsamples,image);
  hipDeviceSynchronize();
}
